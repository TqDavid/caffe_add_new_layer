#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/feat_reshape_common_layer.hpp"

namespace caffe {

// forward: fill zero
template <typename Dtype>
__global__ void ZeroFilling(const int n, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {

    out[index] = 0.0;

  }
}

// forward: copy bottom data to buffer data
template <typename Dtype>
__global__ void PixelCopying(const int n, const Dtype* in, Dtype* out,
    int src_num, int src_channels, int src_height, int src_width,
    int dst_num, int dst_channels, int dst_height, int dst_width) {
  CUDA_KERNEL_LOOP(index, n) {

    // mapping to the src data domain
    int spb = index / (src_width * src_height * src_channels);
    int spc = (index - spb * src_width * src_height * src_channels) / (src_width * src_height);
    int spy = (index - spb * src_width * src_height * src_channels - spc * src_width * src_height) / src_width;
    int spx = (index - spb * src_width * src_height * src_channels - spc * src_width * src_height - spy * src_width) % src_width;

    // mapping to the dst data domain
    int dpx = spx;
    int dpy = spy;
    int dpc = spc;
    int dpb = spb;
    int dst_index = dpx + dpy * dst_width + dpc * dst_width * dst_height + dpb * dst_width * dst_height * dst_channels;
    
    out[dst_index] = in[index];

  }
}

// backward: copy buffer diff to bottom diff
template <typename Dtype>
__global__ void PixelDiffCopying(const int n, const Dtype* in, Dtype* out,
    int src_num, int src_channels, int src_height, int src_width,
    int dst_num, int dst_channels, int dst_height, int dst_width) {
  CUDA_KERNEL_LOOP(index, n) {

    // mapping to the dst diff domain
    int dpb = index / (dst_width * dst_height * dst_channels);
    int dpc = (index - dpb * dst_width * dst_height * dst_channels) / (dst_width * dst_height);
    int dpy = (index - dpb * dst_width * dst_height * dst_channels - dpc * dst_width * dst_height) / dst_width;
    int dpx = (index - dpb * dst_width * dst_height * dst_channels - dpc * dst_width * dst_height - dpy * dst_width) % dst_width;

    // mapping to the src diff domain
    int spx = dpx;
    int spy = dpy;
    int spc = dpc;
    int spb = dpb;
    int src_index = spx + spy * src_width + spc * src_width * src_height + spb * src_width * src_height * src_channels;
    
    // copy
    out[index] = in[src_index];

  }
}

// forward: reshaping the bottom data to top data
template <typename Dtype>
__global__ void PixelReshaping(const int n, const Dtype* in, Dtype* out,
    int src_num, int src_channels, int src_height, int src_width, 
    int dst_num, int dst_channels, int dst_height, int dst_width,
    int sample_step_) {
  CUDA_KERNEL_LOOP(index, n) {
    
    // pixel location decoding in the dst domain
    int dpb = index / (dst_width * dst_height * dst_channels);
    int dpc = (index - dpb * dst_width * dst_height * dst_channels) / (dst_width * dst_height);
    int dpy = (index - dpb * dst_width * dst_height * dst_channels - dpc * dst_width * dst_height) / dst_width;
    int dpx = (index - dpb * dst_width * dst_height * dst_channels - dpc * dst_width * dst_height - dpy * dst_width) % dst_width;

    // pixel location encoding in the src domain
    int spb = dpb;
    int spc = dpc / (sample_step_ * sample_step_);// 4 modify this 
    int block_shift = dpc % (sample_step_ * sample_step_);//4
    int spx = dpx * sample_step_;//2
    int spy = dpy * sample_step_;//2
    if (block_shift == 0) {
      spx += 0;
      spy += 0;
    }else if (block_shift == 1) {
      spx += 1;
      spy += 0;
    }else if (block_shift == 2) {
      spx += 0;
      spy += 1;
    }else {
      spx += 1;
      spy += 1;
    }
    int src_index = spx + spy * src_width + spc * src_width * src_height + spb * src_width * src_height * src_channels; 
    
    // copy
    out[index] = in[src_index];

  }
}

// backward: reshaping the top diff to buffer diff
template <typename Dtype>
__global__ void PixelDiffReshaping(const int n, const Dtype* in, Dtype* out,
    int src_num, int src_channels, int src_height, int src_width,
    int dst_num, int dst_channels, int dst_height, int dst_width,
    int sample_step_) {
  CUDA_KERNEL_LOOP(index, n) {

    // pixel location decoding in the src domain
    int spb = index / (src_width * src_height * src_channels);
    int spc = (index - spb * src_width * src_height * src_channels) / (src_width * src_height);
    int spy = (index - spb * src_width * src_height * src_channels - spc * src_width * src_height) / src_width;
    int spx = (index - spb * src_width * src_height * src_channels - spc * src_width * src_height - spy * src_width) % src_width;
    
    // pixel location encoding in the dst domain
    int dpb = spb;
    int dpc = spc / (sample_step_ * sample_step_); //4 modify this by dtq 20180712
    int block_shift = spc % (sample_step_ * sample_step_);//
    int dpx = spx * sample_step_; //2
    int dpy = spy * sample_step_;//2
    if (block_shift == 0) {
      dpx += 0;
      dpy += 0;
    }else if (block_shift == 1) {
      dpx += 1;
      dpy += 0;
    }else if (block_shift == 2) {
      dpx += 0;
      dpy += 1;
    }else {
      dpx += 1;
      dpy += 1;
    }
    int dst_index = dpx + dpy * dst_width + dpc * dst_width * dst_height + dpb * dst_width * dst_height * dst_channels;

    // copy
    out[dst_index] = in[index];

  }
}

// do forward
template <typename Dtype>
void FeatReshapeCommonLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  //sample_step_ = this->layer_param_.feat_reshape_common_param().sample_step();
  int src_num = bottom[0]->num();
  int src_channels = bottom[0]->channels();
  int src_height = bottom[0]->height();
  int src_width = bottom[0]->width();

  int dst_height = top[0]->height();
  int dst_width = top[0]->width();
  int dst_channels = top[0]->channels();
  int dst_num = top[0]->num();

  int buf_num = src_num;
  int buf_channels = src_channels;
  int buf_height = src_height;
  int buf_width = src_width;

  // if odd add one
  if (src_height % 2 == 1) {
    buf_height += 1;
  }
  if (src_width % 2 == 1) {
    buf_width += 1;
  }
  
  /*LOG(INFO) <<buf_num<<" "<<buf_channels<<" "<<buf_height<<" "<<buf_width;
  LOG(INFO) <<src_num<<" "<<src_channels<<" "<<src_height<<" "<<src_width;
  LOG(INFO) <<this->imgdata_buf_->num()<<" "<<this->imgdata_buf_->channels()<<" "<<this->imgdata_buf_->height()<<" "<<this->imgdata_buf_->width();*/
 

  const int buf_count = this->feat_common_buf_->count();
  const int src_count = bottom[0]->count();
  const int dst_count = top[0]->count();
 
  // filling zero to data buffer
  Dtype* feat_common_buf = this->feat_common_buf_->mutable_gpu_data();
  ZeroFilling<Dtype><<<CAFFE_GET_BLOCKS(buf_count), CAFFE_CUDA_NUM_THREADS>>>(
      buf_count, feat_common_buf);
  CUDA_POST_KERNEL_CHECK;

  // copy src data to buffer data
  const Dtype* bottom_data = bottom[0]->gpu_data();
  feat_common_buf = this->feat_common_buf_->mutable_gpu_data();
  PixelCopying<Dtype><<<CAFFE_GET_BLOCKS(src_count), CAFFE_CUDA_NUM_THREADS>>>(
      src_count, bottom_data, feat_common_buf, 
      src_num, src_channels, src_height, src_width,
      buf_num, buf_channels, buf_height, buf_width);
  CUDA_POST_KERNEL_CHECK;
  
  // pixel reshape
  const Dtype* feat_rbuf = this->feat_common_buf_->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  PixelReshaping<Dtype><<<CAFFE_GET_BLOCKS(dst_count), CAFFE_CUDA_NUM_THREADS>>>(
      dst_count, feat_rbuf, top_data, 
      buf_num, buf_channels, buf_height, buf_width, 
      dst_num, dst_channels, dst_height, dst_width,
      sample_step_);//sample_step_ this param is get from train.prototxt
  CUDA_POST_KERNEL_CHECK;
}

// do backward
template <typename Dtype>
void FeatReshapeCommonLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  //sample_step_ = this->layer_param_.feat_reshape_common_param().sample_step();//we can get this parameter from feat_reshape_common_layer.hpp
  int dst_num = bottom[0]->num();
  int dst_channels = bottom[0]->channels();
  int dst_height = bottom[0]->height();
  int dst_width = bottom[0]->width();

  int src_height = top[0]->height();
  int src_width = top[0]->width();
  int src_channels = top[0]->channels();
  int src_num = top[0]->num();

  int buf_num = dst_num;
  int buf_channels = dst_channels;
  int buf_height = dst_height;
  int buf_width = dst_width;

  // if odd add one
  if (dst_height % 2 == 1) {
    buf_height += 1;
  }
  if (dst_width % 2 == 1) {
    buf_width += 1;
  }

  const int buf_count = this->feat_common_buf_->count();
  const int dst_count = bottom[0]->count();
  const int src_count = top[0]->count();

  // pixel diff reshape
  Dtype* feat_diff_buf = this->feat_common_buf_->mutable_gpu_diff();
  const Dtype* top_diff = top[0]->gpu_diff();
  PixelDiffReshaping<Dtype><<<CAFFE_GET_BLOCKS(src_count), CAFFE_CUDA_NUM_THREADS>>>(
      src_count, top_diff, feat_diff_buf,
      src_num, src_channels, src_height, src_width,
      buf_num, buf_channels, buf_height, buf_width,
      sample_step_);
  CUDA_POST_KERNEL_CHECK;

  // pixel diff copy
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  const Dtype* feat_rbuf = this->feat_common_buf_->gpu_diff();
  PixelDiffCopying<Dtype><<<CAFFE_GET_BLOCKS(dst_count), CAFFE_CUDA_NUM_THREADS>>>(
      dst_count, feat_rbuf, bottom_diff,
      buf_num, buf_channels, buf_height, buf_width,
      dst_num, dst_channels, dst_height, dst_width);
  CUDA_POST_KERNEL_CHECK;

}

INSTANTIATE_LAYER_GPU_FUNCS(FeatReshapeCommonLayer);

}  // namespace caffe
