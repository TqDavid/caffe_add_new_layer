#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/feat_reshape_layer.hpp"

namespace caffe {

// forward: fill zero
template <typename Dtype>
__global__ void ZeroFilling(const int n, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {

    out[index] = 0.0;

  }
}

// forward: copy bottom data to buffer data
template <typename Dtype>
__global__ void PixelCopying(const int n, const Dtype* in, Dtype* out,
    int src_num, int src_channels, int src_height, int src_width,
    int dst_num, int dst_channels, int dst_height, int dst_width) {
  CUDA_KERNEL_LOOP(index, n) {

    // mapping to the src data domain
    int spb = index / (src_width * src_height * src_channels);
    int spc = (index - spb * src_width * src_height * src_channels) / (src_width * src_height);
    int spy = (index - spb * src_width * src_height * src_channels - spc * src_width * src_height) / src_width;
    int spx = (index - spb * src_width * src_height * src_channels - spc * src_width * src_height - spy * src_width) % src_width;

    // mapping to the dst data domain
    int dpx = spx;
    int dpy = spy;
    int dpc = spc;
    int dpb = spb;
    int dst_index = dpx + dpy * dst_width + dpc * dst_width * dst_height + dpb * dst_width * dst_height * dst_channels;
    
    out[dst_index] = in[index];

  }
}

// backward: copy buffer diff to bottom diff
template <typename Dtype>
__global__ void PixelDiffCopying(const int n, const Dtype* in, Dtype* out,
    int src_num, int src_channels, int src_height, int src_width,
    int dst_num, int dst_channels, int dst_height, int dst_width) {
  CUDA_KERNEL_LOOP(index, n) {

    // mapping to the dst diff domain
    int dpb = index / (dst_width * dst_height * dst_channels);
    int dpc = (index - dpb * dst_width * dst_height * dst_channels) / (dst_width * dst_height);
    int dpy = (index - dpb * dst_width * dst_height * dst_channels - dpc * dst_width * dst_height) / dst_width;
    int dpx = (index - dpb * dst_width * dst_height * dst_channels - dpc * dst_width * dst_height - dpy * dst_width) % dst_width;

    // mapping to the src diff domain
    int spx = dpx;
    int spy = dpy;
    int spc = dpc;
    int spb = dpb;
    int src_index = spx + spy * src_width + spc * src_width * src_height + spb * src_width * src_height * src_channels;
    
    // copy
    out[index] = in[src_index];

  }
}

// forward: reshaping the bottom data to top data
template <typename Dtype>
__global__ void PixelReshaping(const int n, const Dtype* in, Dtype* out,
    int src_num, int src_channels, int src_height, int src_width, 
    int dst_num, int dst_channels, int dst_height, int dst_width) {
  CUDA_KERNEL_LOOP(index, n) {
    
    // pixel location decoding in the dst domain
    int dpb = index / (dst_width * dst_height * dst_channels);
    int dpc = (index - dpb * dst_width * dst_height * dst_channels) / (dst_width * dst_height);
    int dpy = (index - dpb * dst_width * dst_height * dst_channels - dpc * dst_width * dst_height) / dst_width;
    int dpx = (index - dpb * dst_width * dst_height * dst_channels - dpc * dst_width * dst_height - dpy * dst_width) % dst_width;

    // pixel location encoding in the src domain
    int spb = dpb;
    int spc = dpc / 4;
    int block_shift = dpc % 4;
    int spx = dpx * 2;
    int spy = dpy * 2;
    if (block_shift == 0) {
      spx += 0;
      spy += 0;
    }else if (block_shift == 1) {
      spx += 1;
      spy += 0;
    }else if (block_shift == 2) {
      spx += 0;
      spy += 1;
    }else {
      spx += 1;
      spy += 1;
    }
    int src_index = spx + spy * src_width + spc * src_width * src_height + spb * src_width * src_height * src_channels; 
    
    // copy
    out[index] = in[src_index];

  }
}

// backward: reshaping the top diff to buffer diff
template <typename Dtype>
__global__ void PixelDiffReshaping(const int n, const Dtype* in, Dtype* out,
    int src_num, int src_channels, int src_height, int src_width,
    int dst_num, int dst_channels, int dst_height, int dst_width) {
  CUDA_KERNEL_LOOP(index, n) {

    // pixel location decoding in the src domain
    int spb = index / (src_width * src_height * src_channels);
    int spc = (index - spb * src_width * src_height * src_channels) / (src_width * src_height);
    int spy = (index - spb * src_width * src_height * src_channels - spc * src_width * src_height) / src_width;
    int spx = (index - spb * src_width * src_height * src_channels - spc * src_width * src_height - spy * src_width) % src_width;

    // pixel location encoding in the dst domain
    int dpb = spb;
    int dpc = spc / 4;
    int block_shift = spc % 4;
    int dpx = spx * 2;
    int dpy = spy * 2;
    if (block_shift == 0) {
      dpx += 0;
      dpy += 0;
    }else if (block_shift == 1) {
      dpx += 1;
      dpy += 0;
    }else if (block_shift == 2) {
      dpx += 0;
      dpy += 1;
    }else {
      dpx += 1;
      dpy += 1;
    }
    int dst_index = dpx + dpy * dst_width + dpc * dst_width * dst_height + dpb * dst_width * dst_height * dst_channels;

    // copy
    out[dst_index] = in[index];

  }
}

// do forward
template <typename Dtype>
void FeatReshapeLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {

  int src_num = bottom[0]->num();
  int src_channels = bottom[0]->channels();
  int src_height = bottom[0]->height();
  int src_width = bottom[0]->width();

  int dst_height = top[0]->height();
  int dst_width = top[0]->width();
  int dst_channels = top[0]->channels();
  int dst_num = top[0]->num();

  int buf_num = src_num;
  int buf_channels = src_channels;
  int buf_height = src_height;
  int buf_width = src_width;

  // if odd add one
  if (src_height % 2 == 1) {
    buf_height += 1;
  }
  if (src_width % 2 == 1) {
    buf_width += 1;
  }
  
  /*LOG(INFO) <<buf_num<<" "<<buf_channels<<" "<<buf_height<<" "<<buf_width;
  LOG(INFO) <<src_num<<" "<<src_channels<<" "<<src_height<<" "<<src_width;
  LOG(INFO) <<this->imgdata_buf_->num()<<" "<<this->imgdata_buf_->channels()<<" "<<this->imgdata_buf_->height()<<" "<<this->imgdata_buf_->width();*/
 

  const int buf_count = this->feat_buf_->count();
  const int src_count = bottom[0]->count();
  const int dst_count = top[0]->count();
 
  // filling zero to data buffer
  Dtype* feat_buf = this->feat_buf_->mutable_gpu_data();
  ZeroFilling<Dtype><<<CAFFE_GET_BLOCKS(buf_count), CAFFE_CUDA_NUM_THREADS>>>(
      buf_count, feat_buf);
  CUDA_POST_KERNEL_CHECK;

  // copy src data to buffer data
  const Dtype* bottom_data = bottom[0]->gpu_data();
  feat_buf = this->feat_buf_->mutable_gpu_data();
  PixelCopying<Dtype><<<CAFFE_GET_BLOCKS(src_count), CAFFE_CUDA_NUM_THREADS>>>(
      src_count, bottom_data, feat_buf, 
      src_num, src_channels, src_height, src_width,
      buf_num, buf_channels, buf_height, buf_width);
  CUDA_POST_KERNEL_CHECK;
  
  // pixel reshape
  const Dtype* feat_rbuf = this->feat_buf_->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  PixelReshaping<Dtype><<<CAFFE_GET_BLOCKS(dst_count), CAFFE_CUDA_NUM_THREADS>>>(
      dst_count, feat_rbuf, top_data, 
      buf_num, buf_channels, buf_height, buf_width, 
      dst_num, dst_channels, dst_height, dst_width);
  CUDA_POST_KERNEL_CHECK;
}

// do backward
template <typename Dtype>
void FeatReshapeLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  
  int dst_num = bottom[0]->num();
  int dst_channels = bottom[0]->channels();
  int dst_height = bottom[0]->height();
  int dst_width = bottom[0]->width();

  int src_height = top[0]->height();
  int src_width = top[0]->width();
  int src_channels = top[0]->channels();
  int src_num = top[0]->num();

  int buf_num = dst_num;
  int buf_channels = dst_channels;
  int buf_height = dst_height;
  int buf_width = dst_width;

  // if odd add one
  if (dst_height % 2 == 1) {
    buf_height += 1;
  }
  if (dst_width % 2 == 1) {
    buf_width += 1;
  }

  const int buf_count = this->feat_buf_->count();
  const int dst_count = bottom[0]->count();
  const int src_count = top[0]->count();

  // pixel diff reshape
  Dtype* feat_diff_buf = this->feat_buf_->mutable_gpu_diff();
  const Dtype* top_diff = top[0]->gpu_diff();
  PixelDiffReshaping<Dtype><<<CAFFE_GET_BLOCKS(src_count), CAFFE_CUDA_NUM_THREADS>>>(
      src_count, top_diff, feat_diff_buf,
      src_num, src_channels, src_height, src_width,
      buf_num, buf_channels, buf_height, buf_width);
  CUDA_POST_KERNEL_CHECK;

  // pixel diff copy
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  const Dtype* feat_rbuf = this->feat_buf_->gpu_diff();
  PixelDiffCopying<Dtype><<<CAFFE_GET_BLOCKS(dst_count), CAFFE_CUDA_NUM_THREADS>>>(
      dst_count, feat_rbuf, bottom_diff,
      buf_num, buf_channels, buf_height, buf_width,
      dst_num, dst_channels, dst_height, dst_width);
  CUDA_POST_KERNEL_CHECK;

}

INSTANTIATE_LAYER_GPU_FUNCS(FeatReshapeLayer);

}  // namespace caffe
